#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <time.h>
#include <thrust/device_vector.h>
#include <vector>

using namespace std;

// Get the timer value.
static void get_time(double* ret)
{
	volatile struct timespec val;
	clock_gettime(CLOCK_REALTIME, (struct timespec*)&val);
	*ret = (double)0.000000001 * val.tv_nsec + val.tv_sec;
}

int main(int argc, char* argv[])
{
	int n = atoi(argv[1]);

	vector<double> A(n * n), B(n * n), C(n * n);

	const double dirandmax = 1.0 / RAND_MAX;
	for (int i = 0; i < n * n; i++)
	{
		A[i] = rand() * dirandmax;
		B[i] = rand() * dirandmax;
		C[i] = rand() * dirandmax;
	}

	const double alpha = rand() * dirandmax;
	const double beta = rand() * dirandmax;

	thrust::device_vector<double> dA = A;
	thrust::device_vector<double> dB = B;
	thrust::device_vector<double> dC = C;

	hipblasHandle_t handle;
	hipblasStatus_t cublasErr = hipblasCreate(&handle);
	if (cublasErr != HIPBLAS_STATUS_SUCCESS)
	{
		printf("Error creating CUBLAS context: err = %d!\n", cublasErr);
		exit(-1);
	}

	const int szbatch = 1000;
	while (1)
	{
		double start; get_time(&start);
		for (int i = 0; i < szbatch; i++)
		{
			hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
				&alpha, thrust::raw_pointer_cast(&dA[0]), n,
				thrust::raw_pointer_cast(&dB[0]), n, &beta,
				thrust::raw_pointer_cast(&dC[0]), n);
			hipError_t err = hipDeviceSynchronize();
			if (err != hipSuccess)
			{
				printf("CUDA error: err = %d\n", err);
				exit(-1);
			}
		}
		double finish; get_time(&finish);

		const double time = finish - start;
		printf("%f GFLOPS\n", (double)n * n * (2 * n + 3) / (1000 * 1000 * 1000 * time) * szbatch);
	}

	hipblasDestroy(handle);

	return 0;
}
